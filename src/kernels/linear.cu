#include "hip/hip_runtime.h"
#include "src/kernels/linear.h"
#include "src/utils/cuda_debug_utils.cuh"


template <typename T>
void launchLinearGemm(TensorWrapper<T> *input,
                      BaseWeight<T> &weight,
                      TensorWrapper<T> *output,
                      cublasWrapper *cublas_wrapper,
                      bool trans_a,
                      bool trans_b)
{
    
    // y = x * w
    // y^T = w^T * x^T
    
    hipblasOperation_t transA = trans_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t transB = trans_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    int Am = weight.shape[1];
    int Ak = weight.shape[0];

    int Bk = input->shape[1];
    int Bn = input->shape[0];

    int Cm = output->shape[1];
    int Cn = output->shape[0];

    // for ctx attn and self attn qkv linear, assume [bs/token nums, qkv h ead num, head size]
    // for gate & up linear, assume weight.shape=[hidden,2*intersize], output.shape=[bs, 2, inter size]
    Cm = output->shape.size() == 3 ? output->shape[1] * output->shape[2] : output->shape[1];
    // for ctx attn output linear
    Bk = input->shape.size() == 3 ? input->shape[1] * input->shape[2] : input->shape[1];

    int lda = Am;
    int ldb = Bk;
    int ldc = Cm;
    
    if (!trans_a && !trans_b)
    {
        LLM_CHECK_WITH_INFO(Ak == Bk, "2nd dim of input MUST = 1st dim of weight");
    }

    cublas_wrapper->Gemm(transA,
                         transB,
                         trans_b ? Ak : Am, // m
                         Cn,                // n, when load real weight, lmhead weight is same as pre embedding, which shape = [vocab, hidden], so here should transpose b
                         Bk,
                         weight.data,  // A, cur_input_len is for context decoder lmhead
                         lda,          // lda
                         input->data,  // B
                         ldb,          // ldb
                         output->data, // C
                         ldc,          // ldc
                         1.0f,
                         0.0f);
#ifdef PRINT_DATA
    print_data<<<1, 1>>>(output->data);
#else
#endif
}

template <typename T>
void launchLinearStridedBatchGemm(TensorWrapper<T> *input1,
                                  TensorWrapper<T> *input2,
                                  TensorWrapper<T> *output,
                                  cublasWrapper *cublas_wrapper,
                                  bool trans_a,
                                  bool trans_b)
{

    hipblasOperation_t transA = trans_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasOperation_t transB = trans_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;

    int Am = input2->shape[3];
    int Ak = input2->shape[2];

    int Bk = input1->shape[3];
    int Bn = input1->shape[2];

    int Cm = output->shape[3];
    int Cn = output->shape[2];

    int lda = Am;
    int ldb = Bk;
    int ldc = Cm;

    int64_t strideA = Am * Ak; // stride should be val after transpose
    int64_t strideB = Bk * Bn;
    int64_t strideC = Cm * Cn;

    int batchCount = input1->shape[0] * input1->shape[1];
    
    cublas_wrapper->stridedBatchedGemm(transA,
                                       transB,
                                       Cm,           // m
                                       Cn,           // n
                                       Bk,           // k
                                       input2->data, // A,[Bk, Bn]=[bs, head num,  head size,max k len]
                                       lda,
                                       strideA,
                                       input1->data, // B [Ak, An]=[bs, head num,  head size,max q len]
                                       ldb,
                                       strideB,
                                       output->data, // C [[bs, head num,  max k len, max q len]
                                       ldc,
                                       strideC,
                                       batchCount,
                                       1.0f,
                                       0.0f);
#ifdef PRINT_DATA
    print_data<<<1, 1>>>(output->data);
#else
#endif
}

template void launchLinearGemm(TensorWrapper<float> *input,
                               BaseWeight<float> &weight,
                               TensorWrapper<float> *output,
                               cublasWrapper *cublas_wrapper,
                               bool trans_a,
                               bool trans_b);

template void launchLinearGemm(TensorWrapper<half> *input,
                               BaseWeight<half> &weight,
                               TensorWrapper<half> *output,
                               cublasWrapper *cublas_wrapper,
                               bool trans_a,
                               bool trans_b);

template void launchLinearStridedBatchGemm(TensorWrapper<float> *input1,
                                           TensorWrapper<float> *input2,
                                           TensorWrapper<float> *output,
                                           cublasWrapper *cublas_wrapper,
                                           bool trans_a,
                                           bool trans_b);

template void launchLinearStridedBatchGemm(TensorWrapper<half> *input1,
                                           TensorWrapper<half> *input2,
                                           TensorWrapper<half> *output,
                                           cublasWrapper *cublas_wrapper,
                                           bool trans_a,
                                           bool trans_b);
