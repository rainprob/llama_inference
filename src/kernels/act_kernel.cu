#include "hip/hip_runtime.h"
#include "src/kernels/act_kernel.h"

#include "src/utils/cuda_debug_utils.cuh"
#include "src/utils/macro.h"

template<typename T>
__device__ __forceinline__ T silu(const T& in) {
  float x = float(in);
  return (T)(x / (1.0f + expf(-x)));
}

template<>
__device__ __forceinline__ half2 silu<half2>(const half2& in) {
  float x = __half2float(in.x);
  float y = __half2float(in.y);
  x = x / (1.0f + expf(-x));
  y = y / (1.0f + expf(-y));

  return make_half2(__float2half(x), __float2half(y));
}



template<typename T>
__global__ void silu_and_mul_kernel(T* out,               // [bs, intermedia size]
                                    const T* input,       // [bs, 2, intermedia size]
                                    const int intermedia_size) 
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  for(; tid < intermedia_size; tid += blockDim.x)
  {
    const T x = input[bid * 2 * intermedia_size + tid];
    const T y = input[bid * 2 * intermedia_size + tid + intermedia_size];
    out[bid * intermedia_size + tid] = silu<T>(x) * y;
  }
}


template<>
__global__ void silu_and_mul_kernel<half>(half* out,               // [bs, intermedia size]
                                          const half* input,       // [bs, 2, intermedia size]
                                          const int intermedia_size)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  using Vec_t = typename Vec<half>::Type;
  int vec_size = Vec<half>::size;

  for(tid = vec_size * tid; tid < intermedia_size; tid += vec_size * blockDim.x)
  {
    const Vec_t x = *reinterpret_cast<Vec_t*>(const_cast<half*>(&input[bid * 2 * intermedia_size + tid]));
    const Vec_t y = *reinterpret_cast<Vec_t*>(const_cast<half*>(&input[bid * 2 * intermedia_size + tid + intermedia_size]));
    
    *reinterpret_cast<Vec_t*>(&out[bid * intermedia_size + tid]) = __hmul2(silu<Vec_t>(x), y);
  }
}

template<typename T>
void launchAct(TensorWrapper<T>* input, TensorWrapper<T>* out) {
    int batch_size = input->shape[0];
    LLM_CHECK(input->shape[1] == 2);
    int intermedia_size = input->shape[2];
    dim3 grid(batch_size);
    dim3 block(256);
    silu_and_mul_kernel<T><<<grid, block>>>(out->data, input->data, intermedia_size);
#ifdef PRINT_DATA
    print_data<<<1, 1>>>(out->data);
#else
#endif
}
// We must instancite the template, if not, will report linking issue
template void launchAct(TensorWrapper<float>* input, TensorWrapper<float>* output);
template void launchAct(TensorWrapper<half>* input, TensorWrapper<half>* output);
