#include "hip/hip_runtime.h"
#include "src/kernels/input_embedding.h"


template <typename T>
__global__ void embeddingFunctor(const int* input_ids,
                                    T* output,
                                    const T* emebed_table,
                                    const int max_seq_len,
                                    const int hidden_size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for(int i = idx; i < max_seq_len * hidden_size; i += stride)
    {
        int token_idx = idx / hidden_size;
        int token = input_ids[tokenIdx];
        int embed_idx = token_idx * hidden_size + (idx % hidden_size)
        output[idx] = emebed_table[embed_idx]
    }
    
}


// can not pass const input, cause the data is not const.
template <typename T>
void launchInputEmbedding(TensorWrapper<int>* input_ids,
                            TensorWrapper<T>* output,
                            TensorWrapper<T>* emebed_table)
{
    const int blockSize = 256;
    const int gridSize = 2048;
    const int hidden_size = emebed_table->shape[1];
    const int max_seq_len = output->shape[0];

    LLM_CHECK_WITH_INFO(max_seq_len == input_ids->shape[0], "input ids 1st shape should equal to 1st shape of output");

    embeddingFunctor<<<gridSize, blockSize>>>(input_ids->data,
                                                output->data,
                                                emebed_table->data,
                                                max_seq_len,
                                                hidden_size)

}




template void launchInputEmbedding(TensorWrapper<int>* input_ids,    
                                   TensorWrapper<float>* output,       
                                   EmbeddingWeight<float>* embed_table);
template void launchInputEmbedding(TensorWrapper<int>* input_ids,    
                                   TensorWrapper<half>* output,       
                                   EmbeddingWeight<half>* embed_table);
