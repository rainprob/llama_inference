#include "hip/hip_runtime.h"
#include "src/kernels/rms_norm.h"



template <typename T>
__device__ T warpReduce(T val)
{
    for(int i = 32 / 2; i > 0; i >>= 1)
    {
        val += __shfl_xor_sync(0xffffffff, val, i);
    }
    return val;
}

template <typename T>
__device__ T blockReduce(T val)
{
    int tid = threadIdx.x;
    int wid = tid / 32;
    int laneid = tid % 32;
    int warpnum = (blockDim.x + 32 - 1) / 32;

    __shared__ T warpsum[32];

    val = warpReduce<T>(val);
    
    if(laneid == 0) warpsum[wid] = val;
    __syncthreads();
    
    T sum = tid < warpnum ? warpsum[wid] : (T)0;
    sum = warpReduce<T>(sum);

    return sum;
}

template <typename T>
__global__ void rmsNormFunctor(T* activation,
                                T* residual,
                                T* weight,
                                float eps,
                                int hidden_state
                                )
{
    using Vec_t = typename Vec<T>::Type;
    int vec_size = Vec<T>::size;

    Vec_t* data = reinterpret_cast<Vec_t*>(activation + blockIdx.x * hidden_state);
    Vec_t* rsd = reinterpret_cast<Vec_t*>(residual + blockIdx.x * hidden_state);
    float thread_sum = 0.0f;

    for(int i = threadIdx.x; i < hidden_state / vec_size; i += blockDim.x)
    {
        
        Vec_t vec = data[i];
        rsd[i] = vec;
        thread_sum += vec.x * vec.x;
        thread_sum += vec.y * vec.y;
        thread_sum += vec.z * vec.z;
        thread_sum += vec.w * vec.w;       
    }

    thread_sum = blockReduce<T>(thread_sum);

    __shared__ float inv_mean;
    
    if(threadIdx.x == 0)    inv_mean = rsqrtf(thread_sum / hidden_state + eps);
    __syncthreads();

    Vec_t* w = reinterpret_cast<Vec_t*>(weight);
   
    for(int i = threadIdx.x; i < hidden_state / vec_size; i += blockDim.x)
    {
        data[i].x = data[i].x * inv_mean * w[i].x;
        data[i].y = data[i].y * inv_mean * w[i].y;
        data[i].z = data[i].z * inv_mean * w[i].z;
        data[i].w = data[i].w * inv_mean * w[i].w;
    }
}

template <>
__global__ void rmsNormFunctor(half* activation,
                                half* residual,
                                half* weight,
                                float eps,
                                int hidden_state
                                )
{
    using Vec_t = typename Vec<half>::Type;
    int vec_size = Vec<half>::size;

    Vec_t* data = reinterpret_cast<Vec_t*>(activation + blockIdx.x * hidden_state);
    Vec_t* rsd = reinterpret_cast<Vec_t*>(residual + blockIdx.x * hidden_state);
    float thread_sum = 0.0f;

    for(int i = threadIdx.x; i < hidden_state / vec_size; i += blockDim.x)
    {
        
        Vec_t vec = data[i];
        rsd[i] = vec;
        thread_sum += __half2float(vec.x) * __half2float(vec.x);
        thread_sum += __half2float(vec.y) * __half2float(vec.y);
    }

    thread_sum = blockReduce<half>(thread_sum);

    __shared__ float inv_mean;
    
    if(threadIdx.x == 0)    inv_mean = rsqrtf(thread_sum / hidden_state + eps);
    __syncthreads();

    Vec_t* w = reinterpret_cast<Vec_t*>(weight);
   
    for(int i = threadIdx.x; i < hidden_state / vec_size; i += blockDim.x)
    { 
        data[i].x = __float2half(__half2float(data[i].x) * inv_mean) * w[i].x;
        data[i].y = __float2half(__half2float(data[i].y) * inv_mean) * w[i].y;
    }
}


template<typename T>
void launchRMSNorm(TensorWrapper<T>* activation,
                    TensorWrapper<T>* residual,
                    RMSNormWeight<T>& weight,
                    float eps,
                    bool is_last)
{
    int hidden_state = activation->shape[1];

    int grid_size = activation->shape[0];
    int block_size = hidden_state / Vec<T>::size;

    rmsNormFunctor<T><<<grid_size, block_size>>>(activation->data,
                                                residual->data,
                                                weight.gamma,
                                                eps,
                                                hidden_state);
    
}

template
void launchRMSNorm(TensorWrapper<float>* activation,
                    TensorWrapper<float>* residual,
                    RMSNormWeight<float>& weight,
                    float eps,
                    bool is_last);

template
void launchRMSNorm(TensorWrapper<half>* activation,
                    TensorWrapper<half>* residual,
                    RMSNormWeight<half>& weight,
                    float eps,
                    bool is_last);